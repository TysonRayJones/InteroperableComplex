#include "hip/hip_runtime.h"

#include <hip/hip_complex.h>

#include "core.both"


// we have to define our own overloads, REEEE

typedef hipDoubleComplex cu_qcomp;

__host__ __device__ inline cu_qcomp operator + (const cu_qcomp& a, const cu_qcomp& b) {
    cu_qcomp res;
    res.x = a.x + b.x;
    res.y = a.y + b.y;
    return res;
}


// here, we attempt to pass 'qcomp' straight to a 'cu_qcomp' argument.
// we expect this won't work; we will have to reinterpret_cast instead

__global__ void kernel_myDoubleFunc(cu_qcomp in, double* d_out){
    *d_out = in.x;
}

double myDoubleFunc(qcomp in) {

    double* d_out;
    hipMalloc(&d_out, sizeof(double));

    cu_qcomp cu_in = reinterpret_cast<cu_qcomp&>(in);
    kernel_myDoubleFunc<<<1, 1>>>(cu_in, d_out);

    double out;
    hipMemcpy(&out, d_out, sizeof(qcomp), hipMemcpyDeviceToHost);
    hipFree(d_out);
    return out;
}


// here, we copy a qcomp straight into a cu_qcomp
// pointer, and back. We expect this to work, because
// qcomp and cu_qcomp have the same memory layout

__global__ void kernel_myCompFunc(cu_qcomp* d_a){
    *d_a = *d_a + *d_a;
}

qcomp myCompFunc(qcomp a) {
    
    cu_qcomp* d_a;
    hipMalloc(&d_a, sizeof(qcomp));
    hipMemcpy(d_a, &a, sizeof(qcomp), hipMemcpyHostToDevice);

    kernel_myCompFunc<<<1, 1>>>(d_a);

    hipMemcpy(&a, d_a, sizeof(qcomp), hipMemcpyDeviceToHost);
    hipFree(d_a);
    return a;
}


// here, we copy a host-memory qcomp-array to a device-memory cu_qcomp-array.
// we expect this to work; it is identical to the above situation, albeit we
// have more than one qcomp to copy

__global__ void kernel_myCompArrFunc(cu_qcomp* d_arr, int len, double* d_out) {
    *d_out = 0;
    for (int i=0; i<len; i++)
        *d_out += d_arr[i].x;
}

// disable name-mangling; this function be directly called by C

extern "C" double myCompArrFunc(qcomp* arr, int len) {

    cu_qcomp* d_arr;
    size_t mem = len * sizeof(qcomp);
    hipMalloc(&d_arr, mem);
    hipMemcpy(d_arr, arr, mem, hipMemcpyHostToDevice);

    double* d_out;
    hipMalloc(&d_out, sizeof(double));

    kernel_myCompArrFunc<<<1, 1>>>(d_arr, len, d_out);

    double out;
    hipMemcpy(&out, d_out, size(double), hipMemcpyDeviceToHost);
    hipFree(d_arr);
    hipFree(d_out);
    return a;
}


// we don't bother making a kernel for this uninteresting function.
// must disable name-mangling; this function be directly called by C

extern "C" int myIntFunc(int a) {
    return 2*a;
}

