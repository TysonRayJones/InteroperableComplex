#include "hip/hip_runtime.h"
/*
 * This is the C++ CUDA backend, to be compiled with nvcc. 
 * Here, qcomp is always resolved to the C++ complex type,
 * but this cannot be used by kernels deployed to the device;
 * instead, we must use NVIDIA's hipComplex types (aliased to
 * cu_qcomp), which  thankfully have the same memory-layout as 
 * std::complex. This means we can pass pointers to qcomp and
 * cu_qcomp interchangeably, and even cast values back and 
 * forth via reinterpret_cast. We disable name-mangling on the
 * functions we intend to be directly callable by a C user.
 */

#include "core.both"


// create cu_qcomp as a device-friendly equivalent of qcomp

#include <hip/hip_complex.h>

typedef hipDoubleComplex cu_qcomp;

// annoyingly, we must define the overloads ourself

__host__ __device__ inline cu_qcomp operator + (const cu_qcomp& a, const cu_qcomp& b) {
    cu_qcomp res;
    res.x = a.x + b.x;
    res.y = a.y + b.y;
    return res;
}


// here, we  pass a 'qcomp' value straight to a 'cu_qcomp' argument,
// leveraging that their memory layouts are equivalent, via reinterpret_cast.
// Because the host function accepts a 'qcomp' directly, it is C++-callable
// only, so we do not disable name-mangling.

__global__ void kernel_myDoubleFunc(cu_qcomp in, double* d_out){
    *d_out = in.x;
}

double myDoubleFunc(qcomp in) {

    double* d_out;
    hipMalloc(&d_out, sizeof(double));

    cu_qcomp cu_in = reinterpret_cast<cu_qcomp&>(in);
    kernel_myDoubleFunc<<<1, 1>>>(cu_in, d_out);

    double out;
    hipMemcpy(&out, d_out, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_out);
    return out;
}


// here, we memcpy a qcomp straight into a cu_qcomp, and back.
// this is fine because of their identical memory layouts.
// Because the host function accepts a 'qcomp' directly, it is C++-callable
// only, so we do not disable name-mangling.

__global__ void kernel_myCompFunc(cu_qcomp* d_a){
    *d_a = *d_a + *d_a;
}

qcomp myCompFunc(qcomp a) {
    
    cu_qcomp* d_a;
    hipMalloc(&d_a, sizeof(qcomp));
    hipMemcpy(d_a, &a, sizeof(qcomp), hipMemcpyHostToDevice);

    kernel_myCompFunc<<<1, 1>>>(d_a);

    hipMemcpy(&a, d_a, sizeof(qcomp), hipMemcpyDeviceToHost);
    hipFree(d_a);
    return a;
}


// here, we copy a host-memory qcomp-array to a device-memory cu_qcomp-array.
// this is equivalent to above, albeit copying multiple values. We disable
// name-mangling on the host code, because it is to be directly callable in C

__global__ void kernel_myCompArrFunc(cu_qcomp* d_arr, int len, double* d_out) {
    *d_out = 0;
    for (int i=0; i<len; i++)
        *d_out += d_arr[i].x;
}

extern "C" double myCompArrFunc(qcomp* arr, int len) {

    cu_qcomp* d_arr;
    size_t mem = len * sizeof(qcomp);
    hipMalloc(&d_arr, mem);
    hipMemcpy(d_arr, arr, mem, hipMemcpyHostToDevice);

    double* d_out;
    hipMalloc(&d_out, sizeof(double));

    kernel_myCompArrFunc<<<1, 1>>>(d_arr, len, d_out);

    double out;
    hipMemcpy(&out, d_out, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_arr);
    hipFree(d_out);
    return out;
}


// we don't bother making a kernel for this uninteresting function.
// must disable name-mangling; this function be directly called by C

extern "C" int myIntFunc(int a) {
    return 2*a;
}

